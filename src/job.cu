#include <private/job.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

void createJob(job_t *job, std::map<std::string, std::string> elements){
    job->base.job_no = stoul(elements["EP"]);
    job->base.arriv_t = stod(elements["ARRIV_T"]);
    job->base.end_time = job->base.start_time = 0;
    job->base.qty = stoul(elements["QTY"]);

    job->r_qt = stod(elements["R_QT"]);
    job->recipe.ul_recipe = 0;
    strncpy(job->recipe.str_recipe, elements["RECIPE"].c_str(), 6);
    job->urgent = stod(elements["URGENT_W"]);
}

__device__ __host__ double jobGetValue(void * _self){
    list_ele_t * self = (list_ele_t *)_self;
    job_t * j = (job_t *)self->ptr_derived_object;
    return *(j->base.os_seq_gene);
}

__device__ __host__ void initJob(job_t *self){
    _list_init(&self->list);
    self->list.ptr_derived_object = self;
    self->list.get_value = jobGetValue;

    job_base_init(&self->base);
    self->base.ptr_derived_object = self;
}

